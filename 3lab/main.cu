#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void initVector(float* vector, float value)
{
	vector[threadIdx.x + blockDim.x * blockIdx.x] = value;
}

int main(int argc, char *argv[])
{
	int blocks = 1024;
	int threads = 1;
	int size_vector = blocks * threads;
	float time;
	float value = 1.0;

	float *dvector, *hvector;

	hipMalloc((void**) &dvector, size_vector * sizeof(float));
	hvector = (float*) calloc(size_vector, sizeof(float));

	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	for (; threads <= 1024; blocks /= 2, threads *= 2, value++) {
		hipEventRecord(start, 0);

		initVector<<<dim3(blocks), dim3(threads)>>>(dvector, value);	
	
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipDeviceSynchronize();

		hipEventElapsedTime(&time, start, stop);

		hipMemcpy(hvector, dvector, size_vector * sizeof(float), hipMemcpyDeviceToHost);

		printf("Time(%d, %d):\t%.8f\n", blocks, threads, time);

		#if 0
		for (int i = 0; i < size_vector; i++) {
			printf("%g ", hvector[i]);
		}
		printf("\n");
		#endif

		time = 0.0;
		initVector<<<dim3(blocks), dim3(threads)>>>(dvector, 0.0);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dvector);
	free(hvector);

	return 0;
}

