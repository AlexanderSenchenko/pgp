#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (1024 * 1024)
#define FULL_DATA_SIZE (N * 20)

__global__ void kernel(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (idx < N) {
		int idx1 = (idx + 1) % 256;
		int idx2 = (idx + 2) % 256;
		float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
		float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
		c[idx] = (as + bs) / 2;
	}
}

__global__ void gSumVector(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N)
		c[idx] = a[idx] * b[idx];
}

__global__ void gScalarMultVect(int *a, int *b, int *c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	if (idx < N)
		c[idx] = a[idx] * b[idx];
}

int main()
{
	hipDeviceProp_t prop;
	int whichDevice;

	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);
   
	if (!prop.deviceOverlap) {
		printf("Device does not support overlapping\n");
		return 0;
	}

	float time1, time2, time3, time4;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int *host_a, *host_b, *host_c;
	hipHostAlloc((void**) &host_a, FULL_DATA_SIZE * sizeof(int),
	hipHostMallocDefault);
	hipHostAlloc((void**) &host_b, FULL_DATA_SIZE * sizeof(int),
	hipHostMallocDefault);
	hipHostAlloc((void**) &host_c, FULL_DATA_SIZE * sizeof(int),
	hipHostMallocDefault);

	int *dev_a, *dev_b, *dev_c;
	hipMalloc((void**) &dev_a, N * sizeof(int));
	hipMalloc((void**) &dev_b, N * sizeof(int));
	hipMalloc((void**) &dev_c, N * sizeof(int));

	int *dev_a0, *dev_a1, *dev_b0, *dev_b1, *dev_c0, *dev_c1;
	hipMalloc((void**) &dev_a0, N * sizeof(int));
	hipMalloc((void**) &dev_a1, N * sizeof(int));
	hipMalloc((void**) &dev_b0, N * sizeof(int));
	hipMalloc((void**) &dev_b1, N * sizeof(int));
	hipMalloc((void**) &dev_c0, N * sizeof(int));
	hipMalloc((void**) &dev_c1, N * sizeof(int));

	hipStream_t stream, stream0, stream1;
	hipStreamCreate(&stream);
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N) {
		hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
		hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);

		kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);

		hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
	}

	hipStreamSynchronize(stream);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time1, start, stop);

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);

		kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);

		hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);

		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

		kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time2, start, stop);

	hipEventRecord(start,0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N * 2) {
		hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

		hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
		kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);

		hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
	}

	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time3, start, stop);

	int *host_a1, *host_b1, *host_c1;
	host_a1 = (int*) calloc(FULL_DATA_SIZE, sizeof(int));
	host_b1 = (int*) calloc(FULL_DATA_SIZE, sizeof(int));
	host_c1 = (int*) calloc(FULL_DATA_SIZE, sizeof(int));

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += N) {
		hipMemcpy(dev_a, host_a1 + i, N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(dev_b, host_b1 + i, N * sizeof(int), hipMemcpyHostToDevice);

		kernel<<<N / 256, 256>>>(dev_a, dev_b, dev_c);

		hipMemcpy(host_c1 + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time4, start, stop);

	printf("%.4f ms\n", time1);
	printf("%.4f ms\n", time2);
	printf("%.4f ms\n", time3);
	printf("%.4f ms\n", time4);

	free(host_a1);
	free(host_b1);
	free(host_c1);

	hipStreamDestroy(stream);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	hipFree(dev_a0);
	hipFree(dev_a1);
	hipFree(dev_b0);
	hipFree(dev_b1);
	hipFree(dev_c0);
	hipFree(dev_c1);

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
