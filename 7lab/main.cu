#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/tuple.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sequence.h>

#define STEPS 1
#define LENGTH 1024
#define U 0.1f
#define H 0.1f
#define TAU 0.2f
#define THREADS_PER_BLOCK 256

__global__ void kernel(float *fn_1, float *fn)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (i == 0)
		fn[i] = fn_1[i] + 1;
	else
		fn[i] = fn_1[i] + (U * TAU / H) * (fn_1[i - 1] - fn_1[i]);
}

struct stepFunctor
{
	__host__ __device__
	float operator()(thrust::tuple<float&, float&> tuple)
	{
		float valueI_1 = thrust::get<0>(tuple);
		float valueI = thrust::get<1>(tuple); 
		return valueI + (U * TAU / H) * (valueI_1 - valueI);
	}
};

struct stepLeftEdgeFunctor
{
	__host__ __device__ float operator()(float fn_1)
	{ return fn_1 + 1; }
};

int main()
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float time;

	float *fn_1;
	float *fn;
	hipMalloc((void**) &fn_1, LENGTH * sizeof(float));
	hipMalloc((void**) &fn, LENGTH * sizeof(float));

	// PURE CUDA
	float *vector = new float[LENGTH];
	for (int i = 0; i < LENGTH; i++)
		vector[i] = i;

	hipMemcpy(fn_1, vector, LENGTH * sizeof(float),
											hipMemcpyHostToDevice);
	hipMemcpy(fn,   vector, LENGTH * sizeof(float),
											hipMemcpyHostToDevice);

#ifdef PRINTS
	float *temp = new float[LENGTH];
	hipMemcpy(temp, fn_1, LENGTH * sizeof(float),
											hipMemcpyDeviceToHost);
	for(int i = 0; i < LENGTH; i++)
		printf("%5.2f\t", temp[i]);
	printf("\n");
#endif

	hipEventRecord(start);
	for (int i = 0; i < STEPS; i++) {
		kernel<<<LENGTH / THREADS_PER_BLOCK,
									THREADS_PER_BLOCK>>>(fn_1, fn);
		hipDeviceSynchronize();
		hipMemcpy(fn_1, fn, LENGTH * sizeof(float),
											hipMemcpyDeviceToDevice);
		
#ifdef PRINTS
		hipMemcpy(temp, fn, LENGTH * sizeof(float),
											hipMemcpyDeviceToHost);
		for(int i = 0; i < LENGTH; i++)
			printf("%5.2f\t", temp[i]);
		printf("\n");
#endif
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Pure cuda: %f ms\n", time);

	// THRUST
	thrust::device_vector<float> vectorFn_1(LENGTH), vectorFn(LENGTH);
	thrust::sequence(vectorFn_1.begin(), vectorFn_1.end());	

#ifdef PRINTS
	thrust::host_vector<float> vectorHost(LENGTH);
	vectorHost = vectorFn_1;
	for(int i = 0; i < LENGTH; i++)
		printf("%5.2f\t", vectorHost[i]);
	printf("\n");
#endif
	
	hipEventRecord(start);
	for (int i = 0; i < STEPS; i++) {
		thrust::transform(
			thrust::make_zip_iterator(
				thrust::make_tuple(vectorFn_1.begin(), 
									vectorFn_1.begin() + 1 )),
			thrust::make_zip_iterator(
				thrust::make_tuple(vectorFn_1.end() - 1,
									vectorFn_1.end() )),
			vectorFn.begin() + 1,
			stepFunctor()
		);
		thrust::transform(vectorFn_1.begin(), vectorFn_1.begin() + 1,
								vectorFn.begin(), stepLeftEdgeFunctor());
		vectorFn_1 = vectorFn;

#ifdef PRINTS
		vectorHost = vectorFn;
		for(int i = 0; i < LENGTH; i++)
			printf("%5.2f\t", vectorHost[i]);
		printf("\n");
#endif
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("Thrust: %f ms\n", time);

#ifdef PRINTS
	delete[] temp;
#endif
	delete[] vector;
	hipFree(fn_1);
	hipFree(fn);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
